#include "hip/hip_runtime.h"
#include "nne.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <cstdlib>

__global__ void nodeCal(float* inList, float* wList, float* outList, int inputNum);
__global__ void nodeLog(float* outputList);
__global__ void nodeLearn(float input, float learnConst, float* wList, float* outList);

Node::Node() : output(0), input(0), localGrad(0) {
	inputWeightList.push_back(0);
}

Node::Node(int inputNum) : output(0), input(0), localGrad(0) {
	inputWeightList.push_back(0);
	for (int i = 0; i < inputNum; i++) {
		inputWeightList.push_back((float)rand() / RAND_MAX);
	}
}

Node::Node(std::vector<float>& inputWeightList, int nodeIndex, int inputWeightLength) : output(0), input(0), localGrad(0) {
	int offset = nodeIndex * inputWeightLength;
	inputWeightList.push_back(0);
	for (int i = 0; i < inputWeightLength; i++) {
		inputWeightList.push_back(inputWeightList[offset + i]);
	}
}

Node::~Node() {}

Layer::Layer(){}

Layer::Layer(int nodeListLength, int inputWeightLength) {
	Node* newNode;
	for (int i = 0; i < nodeListLength; i++) {
		newNode = new Node(inputWeightLength);
		nodeList.push_back(newNode);
	}
}

Layer::Layer(std::vector<float>& inputWeightList, int nodeListLength, int inputWeightLength) {
	Node* newNode;
	for (int i = 0; i < nodeListLength; i++) {
		newNode = new Node(inputWeightList, i, inputWeightLength);
		nodeList.push_back(newNode);
	}
}

Layer::~Layer() {
	int length = nodeList.size();
	for (int i = 0; i < length; i++) {
		delete nodeList[i];
	}
}

void Layer::forwardCal(std::vector<float>& inputList) {
	int inputNum = inputList.size();
	int outputNum = nodeList.size();
	std::vector<float> weightList;
	float* outputList = new float[outputNum];
	float *dInputList, *dWeightList, *dOutputList;

	hipMalloc(&dInputList, inputNum * sizeof(float));
	hipMalloc(&dWeightList, inputNum * outputNum * sizeof(float));
	hipMalloc(&dOutputList, outputNum * sizeof(float));

	hipMemcpy(dInputList, inputList.data(), inputNum * sizeof(float), hipMemcpyHostToDevice);

	for (int i = 0; i < outputNum; i++) {
		weightList.insert(weightList.end(), nodeList[i]->inputWeightList.begin(), nodeList[i]->inputWeightList.end());
	}

	hipMemcpy(dWeightList, weightList.data(), inputNum * outputNum * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dOutputList, outputList, outputNum * sizeof(float), hipMemcpyHostToDevice);

	nodeCal <<<outputNum, inputNum, sizeof(float) * inputNum>>> (dInputList, dWeightList, dOutputList, inputNum);
	hipMemcpy(outputList, dOutputList, outputNum * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < outputNum; i++) {
		nodeList[i]->input = outputList[i];
	}

	nodeLog <<<1, outputNum >>> (dOutputList);
	hipMemcpy(outputList, dOutputList, outputNum * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dInputList);
	hipFree(dWeightList);
	hipFree(dOutputList);
	for (int i = 0; i < outputNum; i++) {
		nodeList[i]->output = outputList[i];
	}
	delete outputList;
}

void Layer::forwardCal(Layer& bLayer){
	std::vector<Node*> &bNodeList = bLayer.nodeList;
	int inputNum = bNodeList.size();
	int outputNum = nodeList.size();
	std::vector<float> inputList;
	std::vector<float> weightList;
	float* outputList = new float[outputNum];
	float *dInputList, *dWeightList, *dOutputList;
	
	hipMalloc(&dInputList, inputNum * sizeof(float));
	hipMalloc(&dWeightList, inputNum * outputNum * sizeof(float));
	hipMalloc(&dOutputList, outputNum * sizeof(float));
	
	inputList.push_back(1);
	for (int i = 0; i < inputNum; i++) {
		inputList.push_back((*bNodeList[i]).output);
	}
	inputNum++;

	hipMemcpy(dInputList, inputList.data(), inputNum * sizeof(float), hipMemcpyHostToDevice);

	for (int i = 0; i < outputNum; i++) {
		weightList.insert(weightList.end(), nodeList[i]->inputWeightList.begin(), nodeList[i]->inputWeightList.end());
	}

	hipMemcpy(dWeightList, weightList.data(), inputNum * outputNum * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dOutputList, outputList, outputNum * sizeof(float), hipMemcpyHostToDevice);

	nodeCal <<<outputNum, inputNum, sizeof(float) * inputNum >>> (dInputList, dWeightList, dOutputList, inputNum);
	hipMemcpy(outputList, dOutputList, outputNum * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < outputNum; i++) {
		nodeList[i]->input = outputList[i];
	}
	nodeLog <<<1, outputNum >>> (dOutputList);
	hipMemcpy(outputList, dOutputList, outputNum * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dInputList);
	hipFree(dWeightList);
	hipFree(dOutputList);
	for (int i = 0; i < outputNum; i++) {
		nodeList[i]->output = outputList[i];
	}
	delete outputList;
}

void Layer::backPropa(Layer& fLayer, float learningFactor) {
	std::vector<Node*> &bNodeList = bLayer.nodeList;
	int inputNum = bNodeList.size();
	int outputNum = nodeList.size();
	std::vector<float> inputList;
	std::vector<float> weightList;
	float* outputList = new float[outputNum];
	float *dInputList, *dWeightList, *dOutputList;

	hipMalloc(&dInputList, inputNum * sizeof(float));
	hipMalloc(&dWeightList, inputNum * outputNum * sizeof(float));
	hipMalloc(&dOutputList, outputNum * sizeof(float));

	inputList.push_back(1);
	for (int i = 0; i < inputNum; i++) {
		inputList.push_back((*bNodeList[i]).output);
	}
	inputNum++;

	hipMemcpy(dInputList, inputList.data(), inputNum * sizeof(float), hipMemcpyHostToDevice);

	for (int i = 0; i < outputNum; i++) {
		weightList.insert(weightList.end(), nodeList[i]->inputWeightList.begin(), nodeList[i]->inputWeightList.end());
	}

	hipMemcpy(dWeightList, weightList.data(), inputNum * outputNum * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dOutputList, outputList, outputNum * sizeof(float), hipMemcpyHostToDevice);
}

__global__ void nodeCal(float* inputList, float* weightList, float* outputList, int inputNum){
	int outputIdx = blockIdx.x * blockDim.x + threadIdx.x;
	float result = 0;
	extern __shared__ float results[];
	results[threadIdx.x] = inputList[threadIdx.x] * weightList[outputIdx];
	__syncthreads();
	for (int i = 0; i < inputNum; i++) {
		result += results[i];
	}
	outputList[blockIdx.x] = result;
	//result = 0;

	//result += subresult;
	//outputList[blockIdx.x] = result;
}

__global__ void nodeLog(float* outputList) {
	outputList[threadIdx.x] = tanh(outputList[threadIdx.x]);
}

__global__ void nodeLearn(float input, float learnConst, float* wList, float* outList) {

}

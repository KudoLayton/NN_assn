#include "hip/hip_runtime.h"
#include "nne.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <cstdlib>

__global__ void nodeCal(float* inList, float* wList, float* outList, int inputNum);
__global__ void nodeLog(float* outputList);
__global__ void nodeGradCal(float* inputList, float* wList, float* outputList, float* gradList, int outputNum);
__global__ void nodeDelLog(float* inputList, float* gradList);
__global__ void nodeLearn(float *inputList, float *delList, float *weightList, float learningFactor, int inputNum);

Node::Node() : output(0), input(0), localGrad(0) {
	inputWeightList.push_back(0);
}

Node::Node(int inputNum) : output(0), input(0), localGrad(0) {
	inputWeightList.push_back(0);
	for (int i = 0; i < inputNum; i++) {
		inputWeightList.push_back((float)rand() / RAND_MAX);
	}
}

Node::Node(std::vector<float>& inputWeightList, int nodeIndex, int inputWeightLength) : output(0), input(0), localGrad(0) {
	int offset = nodeIndex * inputWeightLength;
	inputWeightList.push_back(0);
	for (int i = 0; i < inputWeightLength; i++) {
		inputWeightList.push_back(inputWeightList[offset + i]);
	}
}

Node::~Node() {}

Layer::Layer(){}

Layer::Layer(int nodeListLength, int inputWeightLength) {
	Node* newNode;
	for (int i = 0; i < nodeListLength; i++) {
		newNode = new Node(inputWeightLength);
		nodeList.push_back(newNode);
	}
}

Layer::Layer(std::vector<float>& inputWeightList, int nodeListLength, int inputWeightLength) {
	Node* newNode;
	for (int i = 0; i < nodeListLength; i++) {
		newNode = new Node(inputWeightList, i, inputWeightLength);
		nodeList.push_back(newNode);
	}
}

Layer::~Layer() {
	int length = nodeList.size();
	for (int i = 0; i < length; i++) {
		delete nodeList[i];
	}
}

void Layer::forwardCal(std::vector<float>& inputList) {
	int inputNum = inputList.size();
	int outputNum = nodeList.size();
	std::vector<float> weightList;
	float* outputList = new float[outputNum];
	float *dInputList, *dWeightList, *dOutputList;

	hipMalloc(&dInputList, inputNum * sizeof(float));
	hipMalloc(&dWeightList, inputNum * outputNum * sizeof(float));
	hipMalloc(&dOutputList, outputNum * sizeof(float));

	hipMemcpy(dInputList, inputList.data(), inputNum * sizeof(float), hipMemcpyHostToDevice);

	for (int i = 0; i < outputNum; i++) {
		weightList.insert(weightList.end(), nodeList[i]->inputWeightList.begin(), nodeList[i]->inputWeightList.end());
	}

	hipMemcpy(dWeightList, weightList.data(), inputNum * outputNum * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dOutputList, outputList, outputNum * sizeof(float), hipMemcpyHostToDevice);

	nodeCal <<<outputNum, inputNum, sizeof(float) * inputNum>>> (dInputList, dWeightList, dOutputList, inputNum);
	hipMemcpy(outputList, dOutputList, outputNum * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < outputNum; i++) {
		nodeList[i]->input = outputList[i];
	}

	nodeLog <<<1, outputNum >>> (dOutputList);
	hipMemcpy(outputList, dOutputList, outputNum * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dInputList);
	hipFree(dWeightList);
	hipFree(dOutputList);
	for (int i = 0; i < outputNum; i++) {
		nodeList[i]->output = outputList[i];
	}
	delete outputList;
}

void Layer::forwardCal(Layer& bLayer){
	std::vector<Node*> &bNodeList = bLayer.nodeList;
	int inputNum = bNodeList.size();
	int outputNum = nodeList.size();
	std::vector<float> inputList;
	std::vector<float> weightList;
	float* outputList = new float[outputNum];
	float *dInputList, *dWeightList, *dOutputList;
	
	hipMalloc(&dInputList, inputNum * sizeof(float));
	hipMalloc(&dWeightList, inputNum * outputNum * sizeof(float));
	hipMalloc(&dOutputList, outputNum * sizeof(float));
	
	inputList.push_back(1);
	for (int i = 0; i < inputNum; i++) {
		inputList.push_back((*bNodeList[i]).output);
	}
	inputNum++;

	hipMemcpy(dInputList, inputList.data(), inputNum * sizeof(float), hipMemcpyHostToDevice);

	for (int i = 0; i < outputNum; i++) {
		weightList.insert(weightList.end(), nodeList[i]->inputWeightList.begin(), nodeList[i]->inputWeightList.end());
	}

	hipMemcpy(dWeightList, weightList.data(), inputNum * outputNum * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dOutputList, outputList, outputNum * sizeof(float), hipMemcpyHostToDevice);

	nodeCal <<<outputNum, inputNum, sizeof(float) * inputNum >>> (dInputList, dWeightList, dOutputList, inputNum);
	hipMemcpy(outputList, dOutputList, outputNum * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < outputNum; i++) {
		nodeList[i]->input = outputList[i];
	}
	nodeLog <<<1, outputNum >>> (dOutputList);
	hipMemcpy(outputList, dOutputList, outputNum * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dInputList);
	hipFree(dWeightList);
	hipFree(dOutputList);
	for (int i = 0; i < outputNum; i++) {
		nodeList[i]->output = outputList[i];
	}
	delete outputList;
}

void Layer::getGrad(Layer& fLayer) {
	int inputNum = nodeList.size();
	int outputNum = fLayer.nodeList.size();
	std::vector<Node*> &fNodeList = fLayer.nodeList;
	std::vector<float> inputList;
	std::vector<float> weightList;
	std::vector<float> outputList;
	float *gradList = new float[inputNum];
	float *dInputList, *dWeightList, *dOutputList, *dGradList;

	hipMalloc(&dInputList, inputNum * sizeof(float));
	hipMalloc(&dWeightList, inputNum * outputNum * sizeof(float));
	hipMalloc(&dOutputList, outputNum * sizeof(float));
	hipMalloc(&dGradList, inputNum * sizeof(float));

	for (int i = 0; i < inputNum; i++) {
		inputList.push_back(nodeList[i]->input);
	}

	hipMemcpy(dInputList, inputList.data(), inputNum * sizeof(float), hipMemcpyHostToDevice);

	for (int i = 0; i < outputNum; i++) {
		outputList.push_back(fNodeList[i]->localGrad);
		weightList.insert(weightList.end(), ++(fNodeList[i]->inputWeightList.begin()), fNodeList[i]->inputWeightList.end());
	}

	hipMemcpy(dOutputList, outputList.data(), outputNum * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dWeightList, weightList.data(), inputNum * outputNum * sizeof(float), hipMemcpyHostToDevice);

	nodeGradCal <<<inputNum, outputNum, sizeof(float) * outputNum >>> (dInputList, dWeightList, dOutputList, dGradList, inputNum);
	nodeDelLog <<<1, inputNum >>> (dInputList, dGradList);
	hipMemcpy(gradList, dGradList, inputNum * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dInputList);
	hipFree(dWeightList);
	hipFree(dOutputList);
	hipFree(dGradList);

	for (int i = 0; i < inputNum; i++) {
		nodeList[i]->localGrad = gradList[i];
	}

	delete gradList;
}

float Layer::getGrad(std::vector<float>& answerList) {
	int inputNum = nodeList.size();
	std::vector<float> inputList;
	std::vector<float> outputList;
	float *gradList = new float[inputNum];
	float *dInputList, *dOutputList, *dGradList, mse = 0;

	hipMalloc(&dInputList, inputNum * sizeof(float));
	hipMalloc(&dOutputList, inputNum * sizeof(float));
	hipMalloc(&dGradList, inputNum * sizeof(float));

	for (int i = 0; i < inputNum; i++) {
		inputList.push_back(nodeList[i]->input);
		outputList.push_back(answerList[i] - nodeList[i]->localGrad);
	}
	memcpy(gradList, outputList.data(), inputNum * sizeof(float));

	hipMemcpy(dInputList, inputList.data(), inputNum * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dOutputList, outputList.data(), inputNum * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gradList, dGradList, inputNum * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < inputNum; i++) {
		nodeList[i]->localGrad = gradList[i];
	}
	hipFree(dInputList);
	hipFree(dOutputList);
	hipFree(dGradList);

	for (int i = 0; i < inputNum; i++) {
		nodeList[i]->localGrad = gradList[i];
	}

	delete gradList;
	for (int i = 0; i < inputNum; i++) {
		mse += outputList[i] * outputList[i];
	}
	mse /= inputNum;
	return mse;
}

void Layer::learnWeight(Layer& bLayer, float learningFactor) {
	std::vector<Node*> &bNodeList = bLayer.nodeList;
	int inputNum = bNodeList.size() + 1;
	int outputNum = nodeList.size();
	std::vector<float> inputList;
	std::vector<float> delList;
	float *weightList = new float[inputNum * outputNum];
	float *dInputList, *dDelList, *dWeightList;
	dim3 threadGrid(inputNum, outputNum);
	hipMalloc(&dInputList, inputNum * sizeof(float));
	hipMalloc(&dDelList, outputNum * sizeof(float));
	hipMalloc(&dWeightList, inputNum * outputNum * sizeof(float));

	inputList.push_back(1);

	for (int i = 0; i < inputNum; i++) {
		inputList.push_back(bNodeList[i]->output);
	}

	hipMemcpy(dInputList, inputList.data(), inputNum * sizeof(float), hipMemcpyHostToDevice);

	for (int i = 0; i < outputNum; i++) {
		memcpy(weightList + i * inputNum, nodeList[i]->inputWeightList.data(), inputNum * sizeof(float));
		delList.push_back(nodeList[i]->localGrad);
	}

	hipMemcpy(dWeightList, weightList, inputNum * outputNum * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dDelList, delList.data(), outputNum * sizeof(float), hipMemcpyHostToDevice);

	nodeLearn <<<1, threadGrid >>> (dInputList, dDelList, dWeightList, learningFactor, inputNum);
}

void Layer::learnWeight(std::vector<float>& inputList, float learningFactor){
	int inputNum = inputList.size() + 1;
	int outputNum = nodeList.size();
	std::vector<float> delList;
	float *weightList = new float[inputNum * outputNum];
	float *dInputList, *dDelList, *dWeightList;
	dim3 threadGrid(inputNum, outputNum);
	hipMalloc(&dInputList, inputNum * sizeof(float));
	hipMalloc(&dDelList, outputNum * sizeof(float));
	hipMalloc(&dWeightList, inputNum * outputNum * sizeof(float));

	inputList.insert(inputList.begin(), 1);

	hipMemcpy(dInputList, inputList.data(), inputNum * sizeof(float), hipMemcpyHostToDevice);

	for (int i = 0; i < outputNum; i++) {
		memcpy(weightList + i * inputNum, nodeList[i]->inputWeightList.data(), inputNum * sizeof(float));
		delList.push_back(nodeList[i]->localGrad);
	}

	hipMemcpy(dWeightList, weightList, inputNum * outputNum * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dDelList, delList.data(), outputNum * sizeof(float), hipMemcpyHostToDevice);

	nodeLearn <<<1, threadGrid >>> (dInputList, dDelList, dWeightList, learningFactor, inputNum);
}

__global__ void nodeCal(float* inputList, float* weightList, float* outputList, int inputNum){
	int outputIdx = blockIdx.x * blockDim.x + threadIdx.x;
	float result = 0;
	extern __shared__ float results[];
	results[threadIdx.x] = inputList[threadIdx.x] * weightList[outputIdx];
	__syncthreads();
	for (int i = 0; i < inputNum; i++) {
		result += results[i];
	}
	outputList[blockIdx.x] = result;
}

__global__ void nodeLog(float* outputList) {
	outputList[threadIdx.x] = tanh(outputList[threadIdx.x]);
}

__global__ void nodeGradCal(float* inputList, float* wList, float* outputList, float* gradList, int outputNum) {
	int weightIdx = blockIdx.x + threadIdx.x * blockDim.x;
	extern __shared__ float results[];
	float result = 0;
	results[threadIdx.x] = outputList[threadIdx.x] * wList[weightIdx];
	__syncthreads();
	for (int i = 0; i < outputNum; i++) {
		result += results[i];
	}
	gradList[blockIdx.x] = result;
}

__global__ void nodeDelLog(float* inputList, float* gradList) {
	float temp;
	temp = cosh(inputList[threadIdx.x]);
	temp *= temp;
	temp = 1 / temp;
	gradList[threadIdx.x] *= temp;
}

__global__ void nodeLearn(float *inputList, float *delList, float *weightList, float learningFactor, int inputNum) {
	int weightIdx = threadIdx.x + threadIdx.y * inputNum;
	weightList[weightIdx] += inputList[threadIdx.x] * delList[threadIdx.y] * learningFactor;
}
